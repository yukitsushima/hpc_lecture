
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void initialize(int *bucket){
  int i = threadIdx.x;
  bucket[i] = 0;
}

__global__ void bucket_add(int *key, int *bucket) {
  int i = threadIdx.x;
  int content = key[i];
  atomicAdd(&bucket[content],1);
}

__global__ void bucket_return(int *key, int num, int offset){
  int i = threadIdx.x;
  key[i+offset] = num;
}

int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key,n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  int *bucket;
  hipMallocManaged(&bucket,range*sizeof(int));
  initialize<<<1,range>>>(bucket);
  //for (int i=0; i<range; i++) {
  //  bucket[i] = 0;
  //}
  hipDeviceSynchronize();
  bucket_add<<<1,n>>>(key,bucket);
  hipDeviceSynchronize();
  //for (int i=0; i<n; i++) {
  //  bucket[key[i]]++;
  //}
  int offset = 0;
  for (int i=0;i<range;i++){
    bucket_return<<<1,bucket[i]>>>(key,i,offset);
    offset += bucket[i];
  }
  hipDeviceSynchronize();
  //for (int i=0, j=0; i<range; i++) {
  //  for (; bucket[i]>0; bucket[i]--) {
  //    key[j++] = i;
  //  }
  //}

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
